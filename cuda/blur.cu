#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLUR_SIZE 3

__global__
void blurKernel(unsigned char *in, unsigned char *out, int w, int h)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  if(row < h && col < w){
    out[row*w + col] = 0
    for(int i=-(BLUR_SIZE)/2; i<= BLUR_SIZE/2; i++){
        for(int j=-(BLUR_SIZE)/2; j<=BLUR_SIZE/2; j++){
            if(row+i < h and col+j < w){
                out[row*w + col] += (unsigned char)in[i*w + j]/(BLUR_SIZE*BLUR_SIZE);
            }
        }
    }
  }
  
}